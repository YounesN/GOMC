
#include <hip/hip_runtime.h>
/*******************************************************************************
GPU OPTIMIZED MONTE CARLO (GOMC) 2.1
Copyright (C) 2016  GOMC Group
A copy of the GNU General Public License can be found in the COPYRIGHT.txt
along with this program, also can be found at <http://www.gnu.org/licenses/>.
********************************************************************************/
#ifdef GOMC_CUDA

#include <cuda.h>
#include "CalculateForceCUDAKernel.cuh"
#include "ConstantDefinitionsCUDAKernel.cuh"
#include "CalculateMinImageCUDAKernel.cuh"
#include "cub/cub.cuh"
#include <stdio.h>

using namespace cub;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(cudaError_t code, const char *file, int line, bool abort=true)
{
   if (code != cudaSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", cudaGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void printFreeMemory()
{
  size_t free_byte ;
  size_t total_byte ;
  cudaError_t cuda_status = cudaMemGetInfo( &free_byte, &total_byte ) ;

  if ( cudaSuccess != cuda_status ){
    printf("Error: cudaMemGetInfo fails, %s \n",
	   cudaGetErrorString(cuda_status) );
    exit(1);
  }
  double free_db = (double)free_byte ;
  double total_db = (double)total_byte ;
  double used_db = total_db - free_db ;
  printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
	 used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}

void CallBoxInterForceGPU(VariablesCUDA *vars,
			  vector<uint> &pair1,
			  vector<uint> &pair2,
			  XYZArray const &currentCoords,
			  XYZArray const &currentCOM,
			  BoxDimensions const &boxAxes,
			  bool electrostatic,
			  vector<double> &particleCharge,
			  vector<int> &particleKind,
			  vector<int> &particleMol,
			  double &rT11,
			  double &rT12,
			  double &rT13,
			  double &rT22,
			  double &rT23,
			  double &rT33,
			  double &vT11,
			  double &vT12,
			  double &vT13,
			  double &vT22,
			  double &vT23,
			  double &vT33,
			  uint const box)
{
  int atomNumber = currentCoords.Count();
  int molNumber = currentCOM.Count();
  int *gpu_pair1, *gpu_pair2;
  int *gpu_particleKind;
  int *gpu_particleMol;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_final_value;

  cudaMalloc((void**) &gpu_pair1, pair1.size() * sizeof(int));
  cudaMalloc((void**) &gpu_pair2, pair2.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleCharge,
	     particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_particleKind, particleKind.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleMol, particleMol.size() * sizeof(int));
  cudaMalloc((void**) &gpu_final_value, sizeof(double));

  cudaMemcpy(gpu_pair1, &pair1[0], pair1.size() * sizeof(int),
		       cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_pair2, &pair2[0], pair2.size() * sizeof(int),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_x, currentCoords.x, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, currentCoords.y, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, currentCoords.z, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comx, currentCOM.x, molNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comy, currentCOM.y, molNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comz, currentCOM.z, molNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
	     particleCharge.size() * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0],
	     particleKind.size() * sizeof(int),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0],
	     particleMol.size() * sizeof(int),
	     cudaMemcpyHostToDevice);

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(pair1.size()/threadsPerBlock) + 1;
  BoxInterForceGPU<<<blocksPerGrid, threadsPerBlock>>>(gpu_pair1,
						       gpu_pair2,
						       vars->gpu_x,
						       vars->gpu_y,
						       vars->gpu_z,
						       vars->gpu_comx,
						       vars->gpu_comy,
						       vars->gpu_comz,
						       boxAxes.GetAxis(box).x,
						       boxAxes.GetAxis(box).y,
						       boxAxes.GetAxis(box).z,
						       electrostatic,
						       gpu_particleCharge,
						       gpu_particleKind,
						       gpu_particleMol,
						       vars->gpu_rT11,
						       vars->gpu_rT12,
						       vars->gpu_rT13,
						       vars->gpu_rT22,
						       vars->gpu_rT23,
						       vars->gpu_rT33,
						       vars->gpu_vT11,
						       vars->gpu_vT12,
						       vars->gpu_vT13,
						       vars->gpu_vT22,
						       vars->gpu_vT23,
						       vars->gpu_vT33,
						       pair1.size(),
						       vars->gpu_sigmaSq,
						       vars->gpu_epsilon_Cn,
						       vars->gpu_n,
						       vars->gpu_VDW_Kind,
						       vars->gpu_isMartini,
						       vars->gpu_count,
						       vars->gpu_rCut,
						       vars->gpu_rCutLow,
						       vars->gpu_rOn,
						       vars->gpu_alpha,
						       vars->gpu_ewald,
						       vars->gpu_diElectric_1);
  cudaDeviceSynchronize();
  // ReduceSum // Virial of LJ
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT11,
		    gpu_final_value, pair1.size());
  cudaMalloc(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT11,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT11, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT12,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT12, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT13,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT13, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT22,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT22, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT23,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT23, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT33,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT33, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);

  if(electrostatic)
  {
    // ReduceSum // Virial of Coulomb
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT11, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT12,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT12, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT13,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT13, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT22,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT22, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT23,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT23, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT33,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT33, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
  }

  cudaFree(d_temp_storage);
  cudaFree(gpu_pair1);
  cudaFree(gpu_pair2);
  cudaFree(gpu_particleKind);
  cudaFree(gpu_particleMol);
  cudaFree(gpu_particleCharge);
  cudaFree(gpu_final_value);
}

void CallForceReciprocalGPU(VariablesCUDA *vars,
			    XYZArray const &currentCoords,
			    XYZArray const &currentCOMDiff,
			    vector<double> &particleCharge,
			    double &rT11,
			    double &rT12,
			    double &rT13,
			    double &rT22,
			    double &rT23,
			    double &rT33,
			    uint imageSize,
			    double constVal,
			    uint box)
{
  int atomNumber = currentCoords.Count();
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_final_value;

  cudaMalloc((void**) &gpu_particleCharge,
	     particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_final_value, sizeof(double));

  cudaMemcpy(vars->gpu_x, currentCoords.x, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, currentCoords.y, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, currentCoords.z, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dx, currentCOMDiff.x, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dy, currentCOMDiff.y, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dz, currentCOMDiff.z, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
	     particleCharge.size() * sizeof(double),
	     cudaMemcpyHostToDevice);

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(imageSize/threadsPerBlock) + 1;
  ForceReciprocalGPU<<<blocksPerGrid,
    threadsPerBlock>>>(vars->gpu_x,
		       vars->gpu_y,
		       vars->gpu_z,
		       vars->gpu_dx,
		       vars->gpu_dy,
		       vars->gpu_dz,
		       vars->gpu_kxRef[box],
		       vars->gpu_kyRef[box],
		       vars->gpu_kzRef[box],
		       vars->gpu_prefactRef[box],
		       vars->gpu_hsqrRef[box],
		       vars->gpu_sumRref[box],
		       vars->gpu_sumIref[box],
		       gpu_particleCharge,
		       vars->gpu_rT11,
		       vars->gpu_rT12,
		       vars->gpu_rT13,
		       vars->gpu_rT22,
		       vars->gpu_rT23,
		       vars->gpu_rT33,
		       constVal,
		       imageSize,
		       atomNumber);

  // ReduceSum // Virial of Reciprocal
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
		    gpu_final_value, imageSize);
  cudaMalloc(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
		    gpu_final_value, imageSize);
  cudaMemcpy(&rT11, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT12,
		    gpu_final_value, imageSize);
  cudaMemcpy(&rT12, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT13,
		    gpu_final_value, imageSize);
  cudaMemcpy(&rT13, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT22,
		    gpu_final_value, imageSize);
  cudaMemcpy(&rT22, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT23,
		    gpu_final_value, imageSize);
  cudaMemcpy(&rT23, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT33,
		    gpu_final_value, imageSize);
  cudaMemcpy(&rT33, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);

  cudaFree(gpu_particleCharge);
  cudaFree(gpu_final_value);
  cudaFree(d_temp_storage);
}

__global__ void BoxInterForceGPU(int *gpu_pair1,
				 int *gpu_pair2,
				 double *gpu_x,
				 double *gpu_y,
				 double *gpu_z,
				 double *gpu_comx,
				 double *gpu_comy,
				 double *gpu_comz,
				 double xAxes,
				 double yAxes,
				 double zAxes,
				 bool electrostatic,
				 double *gpu_particleCharge,
				 int *gpu_particleKind,
				 int *gpu_particleMol,
				 double *gpu_rT11,
				 double *gpu_rT12,
				 double *gpu_rT13,
				 double *gpu_rT22,
				 double *gpu_rT23,
				 double *gpu_rT33,
				 double *gpu_vT11,
				 double *gpu_vT12,
				 double *gpu_vT13,
				 double *gpu_vT22,
				 double *gpu_vT23,
				 double *gpu_vT33,
				 int pairSize,
				 double *gpu_sigmaSq,
				 double *gpu_epsilon_Cn,
				 double *gpu_n,
				 int *gpu_VDW_Kind,
				 int *gpu_isMartini,
				 int *gpu_count,
				 double *gpu_rCut,
				 double *gpu_rCutLow,
				 double *gpu_rOn,
				 double *gpu_alpha,
				 int *gpu_ewald,
				 double *gpu_diElectric_1)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= pairSize)
    return;

  double distSq;
  double virX, virY, virZ;
  double pRF = 0.0, qi_qj, pVF = 0.0;
  //tensors for VDW and real part of electrostatic
  gpu_vT11[threadID] = 0.0, gpu_vT22[threadID] = 0.0, gpu_vT33[threadID] = 0.0;
  gpu_rT11[threadID] = 0.0, gpu_rT22[threadID] = 0.0, gpu_rT33[threadID] = 0.0;
  // extra tensors reserved for later on
  gpu_vT12[threadID] = 0.0, gpu_vT13[threadID] = 0.0, gpu_vT23[threadID] = 0.0;
  gpu_rT12[threadID] = 0.0, gpu_rT13[threadID] = 0.0, gpu_rT23[threadID] = 0.0;
  double diff_comx, diff_comy, diff_comz;

  if(InRcutGPU(distSq, virX, virY, virZ, gpu_x[gpu_pair1[threadID]],
	       gpu_y[gpu_pair1[threadID]], gpu_z[gpu_pair1[threadID]],
	       gpu_x[gpu_pair2[threadID]], gpu_y[gpu_pair2[threadID]],
	       gpu_z[gpu_pair2[threadID]], xAxes, yAxes, zAxes, xAxes/2.0,
	       yAxes/2.0, zAxes/2.0, gpu_rCut[0]))
  {
    diff_comx = gpu_comx[gpu_particleMol[gpu_pair1[threadID]]] -
      gpu_comx[gpu_particleMol[gpu_pair2[threadID]]];
    diff_comy = gpu_comy[gpu_particleMol[gpu_pair1[threadID]]] -
      gpu_comy[gpu_particleMol[gpu_pair2[threadID]]];
    diff_comz = gpu_comz[gpu_particleMol[gpu_pair1[threadID]]] -
      gpu_comz[gpu_particleMol[gpu_pair2[threadID]]];

    diff_comx = MinImageSignedGPU(diff_comx, xAxes, xAxes/2.0);
    diff_comy = MinImageSignedGPU(diff_comy, yAxes, yAxes/2.0);
    diff_comz = MinImageSignedGPU(diff_comz, zAxes, zAxes/2.0);

    if(electrostatic)
    {
      qi_qj = gpu_particleCharge[gpu_pair1[threadID]] *
	gpu_particleCharge[gpu_pair2[threadID]];
      pRF = CalcCoulombForceGPU(distSq, qi_qj, gpu_VDW_Kind[0], gpu_ewald[0],
				gpu_isMartini[0], gpu_alpha[0], gpu_rCut[0],
				gpu_diElectric_1[0]);

      gpu_rT11[threadID] = pRF * (virX * diff_comx);
      gpu_rT22[threadID] = pRF * (virY * diff_comy);
      gpu_rT33[threadID] = pRF * (virZ * diff_comz);

      //extra tensor calculations
      gpu_rT12[threadID] = pRF * (0.5 * (virX * diff_comy + virY * diff_comx));
      gpu_rT13[threadID] = pRF * (0.5 * (virX * diff_comz + virZ * diff_comx));
      gpu_rT23[threadID] = pRF * (0.5 * (virY * diff_comz + virZ * diff_comy));
    }

    pVF = CalcEnForceGPU(distSq, gpu_particleKind[gpu_pair1[threadID]],
			 gpu_particleKind[gpu_pair2[threadID]],
			 gpu_sigmaSq, gpu_n, gpu_epsilon_Cn, gpu_rCut[0],
			 gpu_rOn[0], gpu_isMartini[0], gpu_VDW_Kind[0],
			 gpu_count[0]);

    gpu_vT11[threadID] = pVF * (virX * diff_comx);
    gpu_vT22[threadID] = pVF * (virY * diff_comy);
    gpu_vT33[threadID] = pVF * (virZ * diff_comz);

    //extra tensor calculations
    gpu_vT12[threadID] = pVF * (0.5 * (virX * diff_comy + virY * diff_comx));
    gpu_vT13[threadID] = pVF * (0.5 * (virX * diff_comz + virZ * diff_comx));
    gpu_vT23[threadID] = pVF * (0.5 * (virY * diff_comz + virZ * diff_comy));
  }
}


__global__ void ForceReciprocalGPU(double *gpu_x,
				   double *gpu_y,
				   double *gpu_z,
				   double *gpu_comDx,
				   double *gpu_comDy,
				   double *gpu_comDz,
				   double *gpu_kxRef,
				   double *gpu_kyRef,
				   double *gpu_kzRef,
				   double *gpu_prefactRef,
				   double *gpu_hsqrRef,
				   double *gpu_sumRref,
				   double *gpu_sumIref,
				   double *gpu_particleCharge,
				   double *gpu_rT11,
				   double *gpu_rT12,
				   double *gpu_rT13,
				   double *gpu_rT22,
				   double *gpu_rT23,
				   double *gpu_rT33,
				   double constVal,
				   uint imageSize,
				   uint atomNumber)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;

  double factor, arg;
  int i;
  factor = gpu_prefactRef[threadID] * (gpu_sumRref[threadID] *
				       gpu_sumRref[threadID] +
				       gpu_sumIref[threadID] *
				       gpu_sumIref[threadID]);
  gpu_rT11[threadID] = factor * (1.0 - 2.0 *
				 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
				 gpu_kxRef[threadID] * gpu_kxRef[threadID]);
  gpu_rT12[threadID] = factor * (-2.0 *
				 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
				 gpu_kxRef[threadID] * gpu_kyRef[threadID]);
  gpu_rT13[threadID] = factor * (-2.0 *
				 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
				 gpu_kxRef[threadID] * gpu_kzRef[threadID]);
  gpu_rT22[threadID] = factor * (1.0 - 2.0 *
				 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
				 gpu_kyRef[threadID] * gpu_kyRef[threadID]);
  gpu_rT23[threadID] = factor * (-2.0 *
				 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
				 gpu_kyRef[threadID] * gpu_kzRef[threadID]);
  gpu_rT33[threadID] = factor * (1.0 - 2.0 *
				 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
				 gpu_kzRef[threadID] * gpu_kzRef[threadID]);

  //Intramolecular part
  for(i = 0; i < atomNumber; i++)
  {
    arg = DotProductGPU(gpu_kxRef[threadID], gpu_kyRef[threadID],
		     gpu_kzRef[threadID], gpu_x[i], gpu_y[i], gpu_z[i]);

    factor = gpu_prefactRef[threadID] * 2.0 *
      (gpu_sumIref[threadID] * cos(arg) - gpu_sumRref[threadID] * sin(arg)) *
      gpu_particleCharge[i];

    gpu_rT11[threadID] += factor * (gpu_kxRef[threadID] * gpu_comDx[i]);
    gpu_rT12[threadID] += factor * 0.5 *(gpu_kxRef[threadID] * gpu_comDy[i] +
					 gpu_kyRef[threadID] * gpu_comDx[i]);
    gpu_rT13[threadID] += factor * 0.5 *(gpu_kxRef[threadID] * gpu_comDz[i] +
					 gpu_kzRef[threadID] * gpu_comDx[i]);
    gpu_rT22[threadID] += factor * (gpu_kyRef[threadID] * gpu_comDy[i]);
    gpu_rT13[threadID] += factor * 0.5 *(gpu_kyRef[threadID] * gpu_comDz[i] +
					 gpu_kzRef[threadID] * gpu_comDy[i]);
    gpu_rT33[threadID] += factor * (gpu_kzRef[threadID] * gpu_comDz[i]);
  }
}

__device__ double CalcCoulombForceGPU(double distSq, double qi_qj,
				      int gpu_VDW_Kind, int gpu_ewald,
				      int gpu_isMartini, double gpu_alpha,
				      double gpu_rCut, double gpu_diElectric_1)
{
  if(gpu_VDW_Kind == GPU_VDW_STD_KIND)
  {
    return CalcCoulombVirParticleGPU(distSq, qi_qj, gpu_alpha);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SHIFT_KIND)
  {
    return CalcCoulombVirShiftGPU(distSq, qi_qj, gpu_ewald, gpu_alpha);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SWITCH_KIND && gpu_isMartini)
  {
    return CalcCoulombVirSwitchMartiniGPU(distSq, qi_qj, gpu_ewald, gpu_alpha,
					  gpu_rCut, gpu_diElectric_1);
  }
  else
    return CalcCoulombVirSwitchGPU(distSq, qi_qj, gpu_ewald, gpu_alpha,
				   gpu_rCut);
}

__device__ double CalcEnForceGPU(double distSq, int kind1, int kind2,
				 double *gpu_sigmaSq, double *gpu_n,
				 double *gpu_epsilon_Cn, double gpu_rCut,
				 double gpu_rOn, int gpu_isMartini,
				 int gpu_VDW_Kind, int gpu_count)
{
  int index = FlatIndexGPU(kind1, kind2, gpu_count);
  if(gpu_VDW_Kind == GPU_VDW_STD_KIND)
  {
    return CalcVirParticleGPU(distSq, index, gpu_sigmaSq, gpu_n,
			      gpu_epsilon_Cn);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SHIFT_KIND)
  {
    return CalcVirShiftGPU(distSq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SWITCH_KIND && gpu_isMartini)
  {
    return CalcVirSwitchMartiniGPU(distSq, index, gpu_sigmaSq, gpu_n,
				   gpu_epsilon_Cn, gpu_rCut, gpu_rOn);
  }
  else
    return CalcVirSwitchGPU(distSq, index, gpu_sigmaSq, gpu_epsilon_Cn, gpu_n,
			    gpu_rCut, gpu_rOn);
}

//ElectroStatic Calculation
//**************************************************************//
__device__ double CalcCoulombVirParticleGPU(double distSq, double qi_qj,
					    double gpu_alpha)
{
  double dist = sqrt(distSq);
  double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
  double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
  double temp = 1.0 - erf(gpu_alpha * dist);
  return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
}

__device__ double CalcCoulombVirShiftGPU(double distSq, double qi_qj,
					 int gpu_ewald, double gpu_alpha)
{
  if(gpu_ewald)
  {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  }
  else
  {
    double dist = sqrt(distSq);
    return qi_qj/(distSq * dist);
  }
}
__device__ double CalcCoulombVirSwitchMartiniGPU(double distSq, double qi_qj,
						 int gpu_ewald,
						 double gpu_alpha,
						 double gpu_rCut,
						 double gpu_diElectric_1)
{
  if(gpu_ewald)
  {
     double dist = sqrt(distSq);
     double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
     double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
     double temp = 1.0 - erf(gpu_alpha * dist);
     return  qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  }
  else
  {
     // in Martini, the Coulomb switching distance is zero, so we will have
     // sqrt(distSq) - rOnCoul =  sqrt(distSq)
     double dist = sqrt(distSq);
     double rij_ronCoul_2 = distSq;
     double rij_ronCoul_3 = dist * distSq;

     double A1 = 1.0 * (-(1.0+4)*gpu_rCut)/(pow(gpu_rCut,1.0+2) *
					   pow(gpu_rCut, 2));
     double B1 = -1.0 * (-(1.0+3)*gpu_rCut)/(pow(gpu_rCut,1.0+2) *
					    pow(gpu_rCut, 3));

     double virCoul = A1/rij_ronCoul_2 + B1/rij_ronCoul_3;
     return qi_qj * gpu_diElectric_1 * ( 1.0/(dist * distSq) + virCoul/dist);
  }
}

__device__ double CalcCoulombVirSwitchGPU(double distSq, double qi_qj,
					  int gpu_ewald, double gpu_alpha,
					  double gpu_rCut)
{
  if(gpu_ewald)
  {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  }
  else
  {
    double rCutSq = gpu_rCut * gpu_rCut;
    double dist = sqrt(distSq);
    double switchVal = distSq/rCutSq - 1.0;
    switchVal *= switchVal;

    double dSwitchVal = 2.0 * (distSq/rCutSq - 1.0) * 2.0 * dist/rCutSq;
    return -1.0 * qi_qj * (dSwitchVal/distSq - switchVal/(distSq * dist));
  }
}

//VDW Calculation
//*****************************************************************//
__device__ double CalcVirParticleGPU(double distSq, int index,
				     double *gpu_sigmaSq, double *gpu_n,
				     double *gpu_epsilon_Cn)
{
  double rNeg2 = 1.0/distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index]/2.0);
  return gpu_epsilon_Cn[index] * 6.0 *
    ((gpu_n[index]/6.0) * repulse - attract) * rNeg2;
}

__device__ double CalcVirShiftGPU(double distSq, int index, double *gpu_sigmaSq,
				  double *gpu_n, double *gpu_epsilon_Cn)
{
  double rNeg2 = 1.0/distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index]/2.0);
  return gpu_epsilon_Cn[index] * 6.0 *
    ((gpu_n[index]/6.0) * repulse - attract) * rNeg2;
}

__device__ double CalcVirSwitchMartiniGPU(double distSq, int index,
					  double *gpu_sigmaSq, double *gpu_n,
					  double *gpu_epsilon_Cn,
					  double gpu_rCut, double gpu_rOn)
{
  double r_1 = 1.0/sqrt(distSq);
  double r_8 = pow(r_1, 8);
  double r_n2 = pow(r_1, gpu_n[index]+2);

  double rij_ron = sqrt(distSq) - gpu_rOn;
  double rij_ron_2 = rij_ron * rij_ron;
  double rij_ron_3 = rij_ron_2 * rij_ron;

  double pn = gpu_n[index];
  double An = pn * ((pn+1)*gpu_rOn - (pn+4)*gpu_rCut)/
    (pow(gpu_rCut, pn+2)*pow(gpu_rCut-gpu_rOn, 2));
  double Bn = -pn * ((pn+1)*gpu_rOn-(pn+3)*gpu_rCut)/
    (pow(gpu_rCut, pn+2)*pow(gpu_rCut-gpu_rOn, 3));

  double sig6 = pow(gpu_sigmaSq[index], 3);
  double sign = pow(gpu_sigmaSq[index], pn/2);

  double A6 = 6.0 * ((6.0+1)*gpu_rOn-(6.0+4)*gpu_rCut)/
    (pow(gpu_rCut,6.0+2)*pow(gpu_rCut-gpu_rOn, 2));
  double B6 = -6.0 * ((6.0+1)*gpu_rOn-(6.0+3)*gpu_rCut)/
    (pow(gpu_rCut,6.0+2)*pow(gpu_rCut-gpu_rOn, 3));

  double dshifttempRep = An * rij_ron_2 + Bn * rij_ron_3;
  double dshifttempAtt = A6 * rij_ron_2 + B6 * rij_ron_3;

  const double dshiftRep = ( distSq > gpu_rOn * gpu_rOn ?
			     dshifttempRep * r_1 : 0);
  const double dshiftAtt = ( distSq > gpu_rOn * gpu_rOn ?
			     dshifttempAtt * r_1 : 0);
  double Wij = gpu_epsilon_Cn[index] * (sign * (pn * r_n2 + dshiftRep) -
					sig6 * (6.0 * r_8 + dshiftAtt));
  return Wij;
}

__device__ double CalcVirSwitchGPU(double distSq, int index,
				   double *gpu_sigmaSq, double *gpu_epsilon_Cn,
				   double *gpu_n, double gpu_rCut,
				   double gpu_rOn)
{
  double rCutSq = gpu_rCut * gpu_rCut;
  double rCutSq_rijSq = rCutSq - distSq;
  double rCutSq_rijSq_Sq = rCutSq_rijSq * rCutSq_rijSq;
  double rOnSq = gpu_rOn * gpu_rOn;

  double rNeg2 = 1.0/distSq;
  double rRat2 = rNeg2 * gpu_sigmaSq[index];
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index]/2.0);
  double factor1 = rCutSq - 3 * rOnSq;
  double factor2 = pow((rCutSq - rOnSq), -3);

  double fE = rCutSq_rijSq_Sq * factor2 * (factor1 + 2 * distSq);
  double fW = 12.0 * factor2 * rCutSq_rijSq * (rOnSq - distSq);

  const double factE = ( distSq > rOnSq ? fE : 1.0);
  const double factW = ( distSq > rOnSq ? fW : 0.0);

  double Wij = gpu_epsilon_Cn[index] * 6.0 *
    ((gpu_n[index]/6.0) * repulse - attract) * rNeg2;
  double Eij = gpu_epsilon_Cn[index] * (repulse - attract);

  return (Wij * factE - Eij * factW);
}

#endif
